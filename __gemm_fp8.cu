#include "hip/hip_runtime.h"
#include <tl_templates/cuda/gemm.h>
#include <tl_templates/cuda/copy.h>
#include <tl_templates/cuda/reduce.h>
#include <tl_templates/cuda/ldsm.h>
#include <tl_templates/cuda/threadblock_swizzle.h>
#include <tl_templates/cuda/debug.h>
#include <hip/hip_fp8.h>
#include <tl_templates/cuda/copy_sm90.h>
#include <tl_templates/cuda/hip/hip_fp8.h>
#include <tl_templates/cuda/gemm_sm90.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" __global__ void main_kernel(__grid_constant__ const CUtensorMap A_desc, __grid_constant__ const CUtensorMap B_desc, fp8_e4_t* __restrict__ C);
extern "C" __global__ void __launch_bounds__(256, 1) main_kernel(__grid_constant__ const CUtensorMap A_desc, __grid_constant__ const CUtensorMap B_desc, fp8_e4_t* __restrict__ C) {
  extern __shared__ __align__(1024) uchar buf_dyn_shmem[];
  float C_local[128];
  __shared__ uint64_t _mbarrier[6];
  if (((int)threadIdx.x) == 0) {
    tl::prefetch_tma_descriptor(A_desc);
    tl::prefetch_tma_descriptor(B_desc);
    tl::mbarrier_init(_mbarrier[0], 128);
    tl::mbarrier_init(_mbarrier[1], 128);
    tl::mbarrier_init(_mbarrier[2], 128);
    tl::mbarrier_init(_mbarrier[3], 128);
    tl::mbarrier_init(_mbarrier[4], 128);
    tl::mbarrier_init(_mbarrier[5], 128);
  }
  __syncthreads();
  if (128 <= ((int)threadIdx.x)) {
    tl::warpgroup_reg_dealloc<24>();
    for (int k = 0; k < 16; ++k) {
      tl::mbarrier_wait(_mbarrier[((k % 3) + 3)], (((k % 6) / 3) ^ 1));
      if (((int)threadIdx.x) == 128) {
        tl::mbarrier_expect_tx(_mbarrier[(k % 3)], 8192);
        tl::tma_load(A_desc, _mbarrier[(k % 3)], (&(((fp8_e4_t*)buf_dyn_shmem)[((k % 3) * 8192)])), (k * 64), (((int)blockIdx.y) * 128));
        tl::mbarrier_expect_tx(_mbarrier[(k % 3)], 8192);
        tl::tma_load(B_desc, _mbarrier[(k % 3)], (&(((fp8_e4_t*)buf_dyn_shmem)[(((k % 3) * 8192) + 24576)])), (k * 64), (((int)blockIdx.x) * 128));
      }
      tl::mbarrier_arrive(_mbarrier[(k % 3)]);
    }
  } else {
    tl::warpgroup_reg_alloc<240>();
    #pragma unroll
    for (int i = 0; i < 64; ++i) {
      *(float2*)(C_local + (i * 2)) = make_float2(0.000000e+00f, 0.000000e+00f);
    }
    tl::fence_proxy_async();
    for (int k_1 = 0; k_1 < 16; ++k_1) {
      tl::mbarrier_wait(_mbarrier[(k_1 % 3)], ((k_1 % 6) / 3));
      tl::gemm_ss<128, 128, 64, 4, 1, 0, 1, 0, true>((&(((fp8_e4_t*)buf_dyn_shmem)[((k_1 % 3) * 8192)])), (&(((fp8_e4_t*)buf_dyn_shmem)[(((k_1 % 3) * 8192) + 24576)])), (&(C_local[0])));
      tl::mbarrier_arrive(_mbarrier[((k_1 % 3) + 3)]);
    }
    #pragma unroll
    for (int i_1 = 0; i_1 < 128; ++i_1) {
      C[(((((((((((int)blockIdx.y) * 131072) + ((i_1 >> 6) * 65536)) + ((((int)threadIdx.x) >> 5) * 16384)) + (((i_1 & 3) >> 1) * 8192)) + (((((int)threadIdx.x) & 31) >> 2) * 1024)) + (((int)blockIdx.x) * 128)) + (((i_1 & 63) >> 2) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + (i_1 & 1))] = ((fp8_e4_t)C_local[i_1]);
    }
  }
}


#define ERROR_BUF_SIZE 1024
static char error_buf[ERROR_BUF_SIZE];

extern "C" const char* get_last_error() {
    return error_buf;
}

extern "C" int init() {
    error_buf[0] = '\0';
    
    hipError_t result_main_kernel = hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 49152);
    if (result_main_kernel != hipSuccess) {
        snprintf(error_buf, ERROR_BUF_SIZE, "Failed to set the allowed dynamic shared memory size to %d with error: %s", 49152, hipGetErrorString(result_main_kernel));
        return -1;
    }

    return 0;
}


extern "C" int call(fp8_e4_t* __restrict__ A, fp8_e4_t* __restrict__ B, fp8_e4_t* __restrict__ C, hipStream_t stream=hipStreamDefault) {

	CUtensorMap A_desc;
	CUtensorMapDataType A_desc_type= (CUtensorMapDataType)0;
	cuuint32_t A_desc_tensorRank= 2;
	void *A_desc_globalAddress= A;
	cuuint64_t A_desc_globalDim[2]= {1024,1024};
	cuuint64_t A_desc_globalStride[2]= {1,1024};
	cuuint32_t A_desc_boxDim[2]= {64,128};
	cuuint32_t A_desc_elementStrides[2]= {1,1};
	CUtensorMapInterleave A_desc_interleave= (CUtensorMapInterleave)0;
	CUtensorMapSwizzle A_desc_swizzle= (CUtensorMapSwizzle)2;
	CUtensorMapL2promotion A_desc_l2Promotion= (CUtensorMapL2promotion)2;
	CUtensorMapFloatOOBfill A_desc_oobFill= (CUtensorMapFloatOOBfill)0;

	hipError_t A_desc_result = CUTLASS_CUDA_DRIVER_WRAPPER_CALL(cuTensorMapEncodeTiled)(
    &A_desc, A_desc_type, A_desc_tensorRank, A_desc_globalAddress, A_desc_globalDim, A_desc_globalStride + 1, A_desc_boxDim, A_desc_elementStrides, A_desc_interleave, A_desc_swizzle, A_desc_l2Promotion, A_desc_oobFill);

	if (A_desc_result != hipSuccess) {
		std::stringstream ss;
		ss << "Error: Failed to initialize the TMA descriptor A_desc";
		snprintf(error_buf, ERROR_BUF_SIZE, "%s", ss.str().c_str());
		return -1;
	}

	CUtensorMap B_desc;
	CUtensorMapDataType B_desc_type= (CUtensorMapDataType)0;
	cuuint32_t B_desc_tensorRank= 2;
	void *B_desc_globalAddress= B;
	cuuint64_t B_desc_globalDim[2]= {1024,1024};
	cuuint64_t B_desc_globalStride[2]= {1,1024};
	cuuint32_t B_desc_boxDim[2]= {64,128};
	cuuint32_t B_desc_elementStrides[2]= {1,1};
	CUtensorMapInterleave B_desc_interleave= (CUtensorMapInterleave)0;
	CUtensorMapSwizzle B_desc_swizzle= (CUtensorMapSwizzle)2;
	CUtensorMapL2promotion B_desc_l2Promotion= (CUtensorMapL2promotion)2;
	CUtensorMapFloatOOBfill B_desc_oobFill= (CUtensorMapFloatOOBfill)0;

	hipError_t B_desc_result = CUTLASS_CUDA_DRIVER_WRAPPER_CALL(cuTensorMapEncodeTiled)(
    &B_desc, B_desc_type, B_desc_tensorRank, B_desc_globalAddress, B_desc_globalDim, B_desc_globalStride + 1, B_desc_boxDim, B_desc_elementStrides, B_desc_interleave, B_desc_swizzle, B_desc_l2Promotion, B_desc_oobFill);

	if (B_desc_result != hipSuccess) {
		std::stringstream ss;
		ss << "Error: Failed to initialize the TMA descriptor B_desc";
		snprintf(error_buf, ERROR_BUF_SIZE, "%s", ss.str().c_str());
		return -1;
	}
	main_kernel<<<dim3(8, 8, 1), dim3(256, 1, 1), 49152, stream>>>(A_desc, B_desc, C);
	TILELANG_CHECK_LAST_ERROR("main_kernel");

	return 0;
}
