#include "hip/hip_runtime.h"
#include <tl_templates/cuda/gemm.h>
#include <tl_templates/cuda/copy.h>
#include <tl_templates/cuda/reduce.h>
#include <tl_templates/cuda/ldsm.h>
#include <tl_templates/cuda/threadblock_swizzle.h>
#include <tl_templates/cuda/debug.h>
#include <hip/hip_fp8.h>
#include <tl_templates/cuda/copy_sm90.h>
#include <tl_templates/cuda/hip/hip_fp8.h>
#include <tl_templates/cuda/gemm_sm90.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


/*
1024 * 1024 * 1024 matmul  fp8 

A shared: [128, 64]
B shared: [64, 128]
C local: [128, 128]

单次 copy 整个shared?  然后 stage 3?

gemm,  transpose_B = True






*/

extern "C" __global__ void main_kernel(__grid_constant__ const CUtensorMap A_desc, __grid_constant__ const CUtensorMap B_desc, fp8_e4_t* __restrict__ C);
extern "C" __global__ void __launch_bounds__(256, 1) main_kernel(__grid_constant__ const CUtensorMap A_desc, __grid_constant__ const CUtensorMap B_desc, fp8_e4_t* __restrict__ C) {
  extern __shared__ __align__(1024) uchar buf_dyn_shmem[]; // 49152 的大小
  float C_local[128];
  __shared__ uint64_t _mbarrier[6];
  // 256个thread,  每个处理 128 * 128 的大小.
  if (((int)threadIdx.x) == 0) {
    tl::prefetch_tma_descriptor(A_desc);
    tl::prefetch_tma_descriptor(B_desc);
    tl::mbarrier_init(_mbarrier[0], 128);
    tl::mbarrier_init(_mbarrier[1], 128);
    tl::mbarrier_init(_mbarrier[2], 128);
    tl::mbarrier_init(_mbarrier[3], 128);
    tl::mbarrier_init(_mbarrier[4], 128);
    tl::mbarrier_init(_mbarrier[5], 128);
  }
  __syncthreads();
  if (128 <= ((int)threadIdx.x)) {
    tl::warpgroup_reg_dealloc<24>();
    for (int k = 0; k < 16; ++k) {
      tl::mbarrier_wait(_mbarrier[((k % 3) + 3)], (((k % 6) / 3) ^ 1));
      if (((int)threadIdx.x) == 128) {
        tl::mbarrier_expect_tx(_mbarrier[(k % 3)], 8192);
        tl::tma_load(A_desc, _mbarrier[(k % 3)], (&(((fp8_e4_t*)buf_dyn_shmem)[((k % 3) * 8192)])), (k * 64), (((int)blockIdx.y) * 128));
        tl::mbarrier_expect_tx(_mbarrier[(k % 3)], 8192);
        tl::tma_load(B_desc, _mbarrier[(k % 3)], (&(((fp8_e4_t*)buf_dyn_shmem)[(((k % 3) * 8192) + 24576)])), (k * 64), (((int)blockIdx.x) * 128));
      }
      tl::mbarrier_arrive(_mbarrier[(k % 3)]);
    }
  } else {
    tl::warpgroup_reg_alloc<240>();
    #pragma unroll
    for (int i = 0; i < 64; ++i) {
      *(float2*)(C_local + (i * 2)) = make_float2(0.000000e+00f, 0.000000e+00f);
    }
    tl::fence_proxy_async();
    for (int k_1 = 0; k_1 < 16; ++k_1) {
      tl::mbarrier_wait(_mbarrier[(k_1 % 3)], ((k_1 % 6) / 3));
      tl::gemm_ss<128, 128, 64, 4, 1, 0, 1, 0, true>((&(((fp8_e4_t*)buf_dyn_shmem)[((k_1 % 3) * 8192)])), (&(((fp8_e4_t*)buf_dyn_shmem)[(((k_1 % 3) * 8192) + 24576)])), (&(C_local[0])));
      tl::mbarrier_arrive(_mbarrier[((k_1 % 3) + 3)]);
    }
    #pragma unroll
    for (int i_1 = 0; i_1 < 128; ++i_1) {
      C[(((((((((((int)blockIdx.y) * 131072) + ((i_1 >> 6) * 65536)) + ((((int)threadIdx.x) >> 5) * 16384)) + (((i_1 & 3) >> 1) * 8192)) + (((((int)threadIdx.x) & 31) >> 2) * 1024)) + (((int)blockIdx.x) * 128)) + (((i_1 & 63) >> 2) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + (i_1 & 1))] = ((fp8_e4_t)C_local[i_1]);
    }
  }
}


#define ERROR_BUF_SIZE 1024
static char error_buf[ERROR_BUF_SIZE];

extern "C" const char* get_last_error() {
    return error_buf;
}

extern "C" int init() {
    error_buf[0] = '\0';
    
    hipError_t result_main_kernel = hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 49152);
    if (result_main_kernel != hipSuccess) {
        snprintf(error_buf, ERROR_BUF_SIZE, "Failed to set the allowed dynamic shared memory size to %d with error: %s", 49152, hipGetErrorString(result_main_kernel));
        return -1;
    }

    return 0;
}


extern "C" int call(fp8_e4_t* __restrict__ A, fp8_e4_t* __restrict__ B, fp8_e4_t* __restrict__ C, hipStream_t stream=hipStreamDefault) {
	CUtensorMap A_desc; // 创建用于A矩阵的张量映射描述符对象
	CUtensorMapDataType A_desc_type = (CUtensorMapDataType)0; // 设置数据类型为CU_TENSOR_MAP_DATA_TYPE_UINT8(值为0)，因为FP8数据在内部表示为无符号8位整数
	cuuint32_t A_desc_tensorRank = 2; // 指定张量维度为2，表示处理二维矩阵(1024×1024)
	void *A_desc_globalAddress = A; // 指向GPU全局内存中矩阵A的起始地址
	cuuint64_t A_desc_globalDim[2] = {1024, 1024}; // 定义张量的完整大小，这里是1024×1024的矩阵
	cuuint64_t A_desc_globalStride[2] = {1, 1024}; // 定义内存布局的步长：第一个维度步长为1(连续元素)，第二个维度步长为1024(每行1024个元素) // A^T
	cuuint32_t A_desc_boxDim[2] = {64, 128}; // 定义TMA一次操作加载的数据块大小：一次加载64×128的数据块，这与共享内存缓冲区大小匹配
	cuuint32_t A_desc_elementStrides[2] = {1, 1}; // 遍历步长，表示连续访问元素(无跳跃)
	CUtensorMapInterleave A_desc_interleave = (CUtensorMapInterleave)0; // 设置为无交错模式(CU_TENSOR_MAP_INTERLEAVE_NONE)
	CUtensorMapSwizzle A_desc_swizzle = (CUtensorMapSwizzle)2; // 使用64字节共享内存银行交错模式(CU_TENSOR_MAP_SWIZZLE_64B)，减少共享内存访问冲突
	CUtensorMapL2promotion A_desc_l2Promotion = (CUtensorMapL2promotion)2; // 设置L2缓存提升策略为128字节(CU_TENSOR_MAP_L2_PROMOTION_L2_128B)，优化从DRAM到L2缓存的数据获取
	CUtensorMapFloatOOBfill A_desc_oobFill = (CUtensorMapFloatOOBfill)0; // 设置越界访问处理为NONE，不特殊处理越界元素

	// 使用cuTensorMapEncodeTiled函数将所有这些参数编码到A_desc中，创建一个硬件可用的TMA描述符
	// TMA是一种高效的内存传输机制，允许GPU以最小的指令开销将数据从全局内存加载到共享内存，专为像矩阵乘法这样的操作优化
	hipError_t A_desc_result = cutlass::call_cuTensorMapEncodeTiled(
		&A_desc, A_desc_type, A_desc_tensorRank, A_desc_globalAddress, A_desc_globalDim, A_desc_globalStride + 1, A_desc_boxDim, A_desc_elementStrides, A_desc_interleave, A_desc_swizzle, A_desc_l2Promotion, A_desc_oobFill);



	if (A_desc_result != hipSuccess) {
		std::stringstream ss;
		ss << "Error: Failed to initialize the TMA descriptor A_desc";
		snprintf(error_buf, ERROR_BUF_SIZE, "%s", ss.str().c_str());
		return -1;
	}

	CUtensorMap B_desc;
	CUtensorMapDataType B_desc_type= (CUtensorMapDataType)0;
	cuuint32_t B_desc_tensorRank= 2;
	void *B_desc_globalAddress= B;
	cuuint64_t B_desc_globalDim[2]= {1024,1024};
	cuuint64_t B_desc_globalStride[2]= {1,1024};
	cuuint32_t B_desc_boxDim[2]= {64,128};
	cuuint32_t B_desc_elementStrides[2]= {1,1};
	CUtensorMapInterleave B_desc_interleave= (CUtensorMapInterleave)0;
	CUtensorMapSwizzle B_desc_swizzle= (CUtensorMapSwizzle)2;
	CUtensorMapL2promotion B_desc_l2Promotion= (CUtensorMapL2promotion)2;
	CUtensorMapFloatOOBfill B_desc_oobFill= (CUtensorMapFloatOOBfill)0;

	hipError_t B_desc_result = CUTLASS_CUDA_DRIVER_WRAPPER_CALL(cuTensorMapEncodeTiled)(
    &B_desc, B_desc_type, B_desc_tensorRank, B_desc_globalAddress, B_desc_globalDim, B_desc_globalStride + 1, B_desc_boxDim, B_desc_elementStrides, B_desc_interleave, B_desc_swizzle, B_desc_l2Promotion, B_desc_oobFill);

	if (B_desc_result != hipSuccess) {
		std::stringstream ss;
		ss << "Error: Failed to initialize the TMA descriptor B_desc";
		snprintf(error_buf, ERROR_BUF_SIZE, "%s", ss.str().c_str());
		return -1;
	}
	main_kernel<<<dim3(8, 8, 1), dim3(256, 1, 1), 49152, stream>>>(A_desc, B_desc, C);
	TILELANG_CHECK_LAST_ERROR("main_kernel");

	return 0;
}
