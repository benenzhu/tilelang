#include "hip/hip_runtime.h"
#include <tl_templates/cuda/gemm.h>
#include <tl_templates/cuda/copy.h>
#include <tl_templates/cuda/reduce.h>
#include <tl_templates/cuda/ldsm.h>
#include <tl_templates/cuda/threadblock_swizzle.h>
#include <tl_templates/cuda/debug.h>

extern "C" __global__ void main_kernel(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C);
extern "C" __global__ void __launch_bounds__(128, 1) main_kernel(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C) {
  extern __shared__ __align__(1024) uchar buf_dyn_shmem[];
  float C_local[128];
  if (((int)blockIdx.z) == 0) {
    #pragma unroll
    for (int i = 0; i < 16; ++i) {
      *(uint4*)(C + (((((((int)blockIdx.y) * 262144) + (i * 16384)) + ((((int)threadIdx.x) >> 4) * 2048)) + (((int)blockIdx.x) * 128)) + ((((int)threadIdx.x) & 15) * 8))) = make_uint4(__pack_half2(half_t(0.000000e+00f), half_t(0.000000e+00f)), __pack_half2(half_t(0.000000e+00f), half_t(0.000000e+00f)), __pack_half2(half_t(0.000000e+00f), half_t(0.000000e+00f)), __pack_half2(half_t(0.000000e+00f), half_t(0.000000e+00f)));
    }
  }
  #pragma unroll
  for (int i_1 = 0; i_1 < 64; ++i_1) {
    *(float2*)(C_local + (i_1 * 2)) = make_float2(0.000000e+00f, 0.000000e+00f);
  }
  for (int ko = 0; ko < 8; ++ko) {
    __syncthreads();
    #pragma unroll
    for (int i_2 = 0; i_2 < 4; ++i_2) {
      *(uint4*)(((half_t*)buf_dyn_shmem) + (((((i_2 * 1024) + ((((int)threadIdx.x) >> 2) * 32)) + (((((((int)threadIdx.x) & 31) >> 4) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 16)) + (((((((int)threadIdx.x) & 15) >> 3) + (((int)threadIdx.x) & 1)) & 1) * 8)) + 4096)) = *(uint4*)(A + ((((((((int)blockIdx.y) * 131072) + (i_2 * 32768)) + ((((int)threadIdx.x) >> 2) * 1024)) + (((int)blockIdx.z) * 256)) + (ko * 32)) + ((((int)threadIdx.x) & 3) * 8)));
    }
    #pragma unroll
    for (int i_3 = 0; i_3 < 4; ++i_3) {
      uint4 condval;
      if ((((int)blockIdx.x) < 8)) {
        condval = *(uint4*)(B + ((((((((int)blockIdx.z) * 262144) + (ko * 32768)) + (i_3 * 8192)) + ((((int)threadIdx.x) >> 4) * 1024)) + (((int)blockIdx.x) * 128)) + ((((int)threadIdx.x) & 15) * 8)));
      } else {
        condval = make_uint4(__pack_half2(half_t(0.000000e+00f), half_t(0.000000e+00f)), __pack_half2(half_t(0.000000e+00f), half_t(0.000000e+00f)), __pack_half2(half_t(0.000000e+00f), half_t(0.000000e+00f)), __pack_half2(half_t(0.000000e+00f), half_t(0.000000e+00f)));
      }
      *(uint4*)(((half_t*)buf_dyn_shmem) + ((((((((((int)threadIdx.x) & 15) >> 3) * 2048) + (i_3 * 512)) + ((((int)threadIdx.x) >> 4) * 64)) + ((((((int)threadIdx.x) >> 6) + ((((int)threadIdx.x) & 7) >> 2)) & 1) * 32)) + (((((((int)threadIdx.x) & 63) >> 5) + ((((int)threadIdx.x) & 3) >> 1)) & 1) * 16)) + (((((((int)threadIdx.x) & 31) >> 4) + (((int)threadIdx.x) & 1)) & 1) * 8))) = condval;
    }
    __syncthreads();
    tl::gemm_ss<128, 128, 32, 2, 2, 0, 0, 0>((&(((half_t*)buf_dyn_shmem)[4096])), (&(((half_t*)buf_dyn_shmem)[0])), (&(C_local[0])));
  }
  #pragma unroll
  for (int i_4 = 0; i_4 < 64; ++i_4) {
    uint1 __1;
    float2 v_ = *(float2*)(C_local + (i_4 * 2));
    ((half2*)(&(__1.x)))->x = (half_t)(v_.x);
    ((half2*)(&(__1.x)))->y = (half_t)(v_.y);
    *(uint1*)(((half_t*)buf_dyn_shmem) + (((((((((i_4 & 7) >> 1) * 4096) + (((((int)threadIdx.x) & 63) >> 5) * 2048)) + ((i_4 & 1) * 1024)) + (((((int)threadIdx.x) & 31) >> 2) * 128)) + ((i_4 >> 3) * 16)) + ((((int)threadIdx.x) >> 6) * 8)) + ((((int)threadIdx.x) & 3) * 2))) = __1;
  }
  __syncthreads();
  #pragma unroll
  for (int i_5 = 0; i_5 < 64; ++i_5) {
    AtomicAddx2((&(C[(((((((int)blockIdx.y) * 262144) + (i_5 * 4096)) + ((((int)threadIdx.x) >> 6) * 2048)) + (((int)blockIdx.x) * 128)) + ((((int)threadIdx.x) & 63) * 2))])), (&(((half_t*)buf_dyn_shmem)[((i_5 * 256) + (((int)threadIdx.x) * 2))])));
  }
}


#define ERROR_BUF_SIZE 1024
static char error_buf[ERROR_BUF_SIZE];

extern "C" const char* get_last_error() {
    return error_buf;
}

extern "C" int init() {
    error_buf[0] = '\0';
    
    hipError_t result_main_kernel = hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 32768);
    if (result_main_kernel != hipSuccess) {
        snprintf(error_buf, ERROR_BUF_SIZE, "Failed to set the allowed dynamic shared memory size to %d with error: %s", 32768, hipGetErrorString(result_main_kernel));
        return -1;
    }

    return 0;
}

extern "C" int call(half_t* __restrict__ A, half_t* __restrict__ B, half_t* __restrict__ C, hipStream_t stream=hipStreamDefault) {
	main_kernel<<<dim3(16, 8, 4), dim3(128, 1, 1), 32768, stream>>>(A, B, C);

    return 0;
}
