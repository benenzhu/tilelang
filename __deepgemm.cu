#include "hip/hip_runtime.h"
#include <tl_templates/cuda/gemm.h>
#include <tl_templates/cuda/copy.h>
#include <tl_templates/cuda/reduce.h>
#include <tl_templates/cuda/ldsm.h>
#include <tl_templates/cuda/threadblock_swizzle.h>
#include <tl_templates/cuda/debug.h>
#include <hip/hip_fp8.h>
#include <tl_templates/cuda/copy_sm90.h>
#include <tl_templates/cuda/hip/hip_fp8.h>
#include <tl_templates/cuda/gemm_sm90.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
extern "C" __global__ void main_kernel(__grid_constant__ const CUtensorMap A_desc, __grid_constant__ const CUtensorMap B_desc, __grid_constant__ const CUtensorMap C_desc, float* __restrict__ scales_a, float* __restrict__ scales_b);
extern "C" __global__ void __launch_bounds__(256, 1) main_kernel(__grid_constant__ const CUtensorMap A_desc, __grid_constant__ const CUtensorMap B_desc, __grid_constant__ const CUtensorMap C_desc, float* __restrict__ scales_a, float* __restrict__ scales_b) {
  extern __shared__ __align__(1024) uchar buf_dyn_shmem[];
  float C_local[128];
  float C_local_accum[128];
  __shared__ uint64_t _mbarrier[18];
  if (((int)threadIdx.x) == 0) {
    tl::prefetch_tma_descriptor(A_desc);
    tl::prefetch_tma_descriptor(B_desc);
    tl::prefetch_tma_descriptor(C_desc);
    tl::mbarrier_init(_mbarrier[0], 128);
    tl::mbarrier_init(_mbarrier[1], 128);
    tl::mbarrier_init(_mbarrier[2], 128);
    tl::mbarrier_init(_mbarrier[3], 128);
    tl::mbarrier_init(_mbarrier[4], 128);
    tl::mbarrier_init(_mbarrier[5], 128);
    tl::mbarrier_init(_mbarrier[6], 128);
    tl::mbarrier_init(_mbarrier[7], 128);
    tl::mbarrier_init(_mbarrier[8], 128);
    tl::mbarrier_init(_mbarrier[9], 128);
    tl::mbarrier_init(_mbarrier[10], 128);
    tl::mbarrier_init(_mbarrier[11], 128);
    tl::mbarrier_init(_mbarrier[12], 128);
    tl::mbarrier_init(_mbarrier[13], 128);
    tl::mbarrier_init(_mbarrier[14], 128);
    tl::mbarrier_init(_mbarrier[15], 128);
    tl::mbarrier_init(_mbarrier[16], 128);
    tl::mbarrier_init(_mbarrier[17], 128);
  }
  __syncthreads();
  if (128 <= ((int)threadIdx.x)) {
    tl::warpgroup_reg_dealloc<24>();
    const dim3 blockIdx = tl::rasterization2DRow<10>();
    for (int k = 0; k < 64; ++k) {
      tl::mbarrier_wait(_mbarrier[((k & 3) + 8)], (((k & 7) >> 2) ^ 1));
      if (((int)threadIdx.x) == 128) {
        tl::mbarrier_expect_tx(_mbarrier[(k & 3)], 16384);
        tl::tma_load(A_desc, _mbarrier[(k & 3)], (&(((fp8_e4_t*)buf_dyn_shmem)[(((k & 3) * 16384) + 67584)])), (k * 128), (((int)blockIdx.y) * 128));
        tl::mbarrier_expect_tx(_mbarrier[(k & 3)], 16384);
        tl::tma_load(A_desc, _mbarrier[(k & 3)], (&(((fp8_e4_t*)buf_dyn_shmem)[(((k & 3) * 16384) + 67584)])), (k * 128), (((int)blockIdx.y) * 128));
        tl::mbarrier_expect_tx(_mbarrier[(k & 3)], 16384);
        tl::tma_load(B_desc, _mbarrier[(k & 3)], (&(((fp8_e4_t*)buf_dyn_shmem)[(((k & 3) * 16384) + 2048)])), (k * 128), (((int)blockIdx.x) * 128));
      }
      tl::mbarrier_arrive(_mbarrier[(k & 3)]);
      tl::mbarrier_wait(_mbarrier[((k & 3) + 12)], (((k & 7) >> 2) ^ 1));
      ((float*)buf_dyn_shmem)[((((k & 3) * 128) + ((int)threadIdx.x)) - 128)] = (scales_a[((((((int)blockIdx.y) * 8192) + (((int)threadIdx.x) * 64)) + k) - 8192)] * scales_b[((((int)blockIdx.x) * 64) + k)]);
      tl::fence_proxy_async();
      tl::mbarrier_cp_async_arrive(_mbarrier[((k & 3) + 4)]);
      tl::mbarrier_arrive(_mbarrier[((k & 3) + 4)]);
    }
  } else {
    tl::warpgroup_reg_alloc<240>();
    const dim3 blockIdx = tl::rasterization2DRow<10>();
    #pragma unroll
    for (int i = 0; i < 64; ++i) {
      *(float2*)(C_local + (i * 2)) = make_float2(0.000000e+00f, 0.000000e+00f);
    }
    #pragma unroll
    for (int i_1 = 0; i_1 < 64; ++i_1) {
      *(float2*)(C_local_accum + (i_1 * 2)) = make_float2(0.000000e+00f, 0.000000e+00f);
    }
    for (int k_1 = 0; k_1 < 64; ++k_1) {
      tl::mbarrier_wait(_mbarrier[(k_1 & 3)], ((k_1 & 7) >> 2));
      tl::gemm_ss<128, 128, 128, 4, 1, 0, 1, 0, true>((&(((fp8_e4_t*)buf_dyn_shmem)[(((k_1 & 3) * 16384) + 67584)])), (&(((fp8_e4_t*)buf_dyn_shmem)[(((k_1 & 3) * 16384) + 2048)])), (&(C_local[0])));
      tl::mbarrier_arrive(_mbarrier[((k_1 & 3) + 8)]);
      tl::mbarrier_wait(_mbarrier[((k_1 & 3) + 4)], ((k_1 & 7) >> 2));
      #pragma unroll
      for (int i_2 = 0; i_2 < 64; ++i_2) {
        float2 __1;
          float2 v_ = *(float2*)(C_local_accum + (i_2 * 2));
          float2 __2;
            float2 v__1 = *(float2*)(C_local + (i_2 * 2));
            float2 v__2 = make_float2(((float*)buf_dyn_shmem)[((((((k_1 & 3) * 128) + ((i_2 >> 5) * 64)) + ((((int)threadIdx.x) >> 5) * 16)) + ((i_2 & 1) * 8)) + ((((int)threadIdx.x) & 31) >> 2))], ((float*)buf_dyn_shmem)[((((((k_1 & 3) * 128) + ((i_2 >> 5) * 64)) + ((((int)threadIdx.x) >> 5) * 16)) + ((i_2 & 1) * 8)) + ((((int)threadIdx.x) & 31) >> 2))]);
            __2.x = (v__1.x*v__2.x);
            __2.y = (v__1.y*v__2.y);
          __1.x = (v_.x+__2.x);
          __1.y = (v_.y+__2.y);
        *(float2*)(C_local_accum + (i_2 * 2)) = __1;
      }
      tl::fence_proxy_async();
      tl::mbarrier_arrive(_mbarrier[((k_1 & 3) + 12)]);
      #pragma unroll
      for (int i_3 = 0; i_3 < 64; ++i_3) {
        *(float2*)(C_local + (i_3 * 2)) = make_float2(0.000000e+00f, 0.000000e+00f);
      }
    }
    tl::syncthreads_partial(_mbarrier[16]);
    #pragma unroll
    for (int i_4 = 0; i_4 < 64; ++i_4) {
      *(float2*)(((float*)buf_dyn_shmem) + ((((((((i_4 >> 5) * 8192) + ((((int)threadIdx.x) >> 5) * 2048)) + ((i_4 & 1) * 1024)) + (((((int)threadIdx.x) & 31) >> 2) * 128)) + (((i_4 & 31) >> 1) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 16896)) = *(float2*)(C_local_accum + (i_4 * 2));
    }
    tl::fence_proxy_async();
    tl::syncthreads_partial(_mbarrier[17]);
    if (((int)threadIdx.x) == 0) {
      tl::tma_store(C_desc, (&(((float*)buf_dyn_shmem)[16896])), (((int)blockIdx.x) * 128), (((int)blockIdx.y) * 128));
      tl::tma_store_arrive();
      tl::tma_store_wait<0>();
    }
  }
}


#define ERROR_BUF_SIZE 1024
static char error_buf[ERROR_BUF_SIZE];

extern "C" const char* get_last_error() {
    return error_buf;
}

extern "C" int init() {
    error_buf[0] = '\0';
    
    hipError_t result_main_kernel = hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 133120);
    if (result_main_kernel != hipSuccess) {
        snprintf(error_buf, ERROR_BUF_SIZE, "Failed to set the allowed dynamic shared memory size to %d with error: %s", 133120, hipGetErrorString(result_main_kernel));
        return -1;
    }

    return 0;
}

extern "C" int call(fp8_e4_t* __restrict__ A, fp8_e4_t* __restrict__ B, float* __restrict__ C, float* __restrict__ scales_a, float* __restrict__ scales_b, hipStream_t stream=hipStreamDefault) {

	CUtensorMap A_desc;
	CUtensorMapDataType A_desc_type= (CUtensorMapDataType)0;
	cuuint32_t A_desc_tensorRank= 2;
	void *A_desc_globalAddress= A;
	cuuint64_t A_desc_globalDim[2]= {8192,1024};
	cuuint64_t A_desc_globalStride[2]= {1,8192};
	cuuint32_t A_desc_boxDim[2]= {128,128};
	cuuint32_t A_desc_elementStrides[2]= {1,1};
	CUtensorMapInterleave A_desc_interleave= (CUtensorMapInterleave)0;
	CUtensorMapSwizzle A_desc_swizzle= (CUtensorMapSwizzle)3;
	CUtensorMapL2promotion A_desc_l2Promotion= (CUtensorMapL2promotion)2;
	CUtensorMapFloatOOBfill A_desc_oobFill= (CUtensorMapFloatOOBfill)0;

	hipError_t A_desc_result = CUTLASS_CUDA_DRIVER_WRAPPER_CALL(cuTensorMapEncodeTiled)(
    &A_desc, A_desc_type, A_desc_tensorRank, A_desc_globalAddress, A_desc_globalDim, A_desc_globalStride + 1, A_desc_boxDim, A_desc_elementStrides, A_desc_interleave, A_desc_swizzle, A_desc_l2Promotion, A_desc_oobFill);

	if (A_desc_result != hipSuccess) {
		std::stringstream ss;
		ss << "Error: Failed to initialize the TMA descriptor A_desc";
		snprintf(error_buf, ERROR_BUF_SIZE, "%s", ss.str().c_str());
		return -1;
	}

	CUtensorMap B_desc;
	CUtensorMapDataType B_desc_type= (CUtensorMapDataType)0;
	cuuint32_t B_desc_tensorRank= 2;
	void *B_desc_globalAddress= B;
	cuuint64_t B_desc_globalDim[2]= {8192,1024};
	cuuint64_t B_desc_globalStride[2]= {1,8192};
	cuuint32_t B_desc_boxDim[2]= {128,128};
	cuuint32_t B_desc_elementStrides[2]= {1,1};
	CUtensorMapInterleave B_desc_interleave= (CUtensorMapInterleave)0;
	CUtensorMapSwizzle B_desc_swizzle= (CUtensorMapSwizzle)3;
	CUtensorMapL2promotion B_desc_l2Promotion= (CUtensorMapL2promotion)2;
	CUtensorMapFloatOOBfill B_desc_oobFill= (CUtensorMapFloatOOBfill)0;

	hipError_t B_desc_result = CUTLASS_CUDA_DRIVER_WRAPPER_CALL(cuTensorMapEncodeTiled)(
    &B_desc, B_desc_type, B_desc_tensorRank, B_desc_globalAddress, B_desc_globalDim, B_desc_globalStride + 1, B_desc_boxDim, B_desc_elementStrides, B_desc_interleave, B_desc_swizzle, B_desc_l2Promotion, B_desc_oobFill);

	if (B_desc_result != hipSuccess) {
		std::stringstream ss;
		ss << "Error: Failed to initialize the TMA descriptor B_desc";
		snprintf(error_buf, ERROR_BUF_SIZE, "%s", ss.str().c_str());
		return -1;
	}

	CUtensorMap C_desc;
	CUtensorMapDataType C_desc_type= (CUtensorMapDataType)7;
	cuuint32_t C_desc_tensorRank= 2;
	void *C_desc_globalAddress= C;
	cuuint64_t C_desc_globalDim[2]= {1024,1024};
	cuuint64_t C_desc_globalStride[2]= {4,4096};
	cuuint32_t C_desc_boxDim[2]= {128,128};
	cuuint32_t C_desc_elementStrides[2]= {1,1};
	CUtensorMapInterleave C_desc_interleave= (CUtensorMapInterleave)0;
	CUtensorMapSwizzle C_desc_swizzle= (CUtensorMapSwizzle)0;
	CUtensorMapL2promotion C_desc_l2Promotion= (CUtensorMapL2promotion)2;
	CUtensorMapFloatOOBfill C_desc_oobFill= (CUtensorMapFloatOOBfill)0;

	hipError_t C_desc_result = CUTLASS_CUDA_DRIVER_WRAPPER_CALL(cuTensorMapEncodeTiled)(
    &C_desc, C_desc_type, C_desc_tensorRank, C_desc_globalAddress, C_desc_globalDim, C_desc_globalStride + 1, C_desc_boxDim, C_desc_elementStrides, C_desc_interleave, C_desc_swizzle, C_desc_l2Promotion, C_desc_oobFill);

	if (C_desc_result != hipSuccess) {
		std::stringstream ss;
		ss << "Error: Failed to initialize the TMA descriptor C_desc";
		snprintf(error_buf, ERROR_BUF_SIZE, "%s", ss.str().c_str());
		return -1;
	}
	main_kernel<<<dim3(8, 8, 1), dim3(256, 1, 1), 133120, stream>>>(A_desc, B_desc, C_desc, scales_a, scales_b);
	TILELANG_CHECK_LAST_ERROR("main_kernel");

	return 0;
}
