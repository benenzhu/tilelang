#include <hip/hip_runtime.h>
#include <iostream>

__global__ void metrics_kernel(float *src, float *dst, int count, int N) {
  int id_x = blockIdx.x * blockDim.x + threadIdx.x;

  if (id_x >= count) {
    return;
  }

  dst[id_x * N] = src[id_x * N];
}

int main(int argc, char *argv[]) {
  int N = 1;

  if (argc == 2) {
    N = atoi(argv[1]);
  }

  const int kCount = 32;

  // alloc N times the memory space for stride load
  int size = kCount * sizeof(float) * N;

  float *src = static_cast<float *>(malloc(size));
  float *dst = static_cast<float *>(malloc(size));

  for (int i = 0; i < kCount * N; ++i) {
    src[i] = i;
  }

  float *src_dev = nullptr;
  float *dst_dev = nullptr;

  hipMalloc(&src_dev, size);
  hipMalloc(&dst_dev, size);

  hipMemcpy(src_dev, src, size, hipMemcpyHostToDevice);

  dim3 block(32, 1);
  dim3 grid(1, 1);

  metrics_kernel<<<grid, block>>>(src_dev, dst_dev, kCount, N);

  hipStreamSynchronize(0);

  hipFree(src_dev);
  hipFree(dst_dev);

  free(src);
  free(dst);

  return 0;
}