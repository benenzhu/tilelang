#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <iomanip>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
#include <numeric> // For std::accumulate
#include <cmath>   // For std::round

// CUDA runtime
#include <hip/hip_runtime.h>
#include <cstdint> // For uint64_t

#define CUDA_CHECK(call)                                                  \
  do {                                                                    \
    hipError_t err = call;                                               \
    if (err != hipSuccess) {                                             \
      fprintf(stderr, "CUDA error at %s %d: %s\n", __FILE__, __LINE__,    \
              hipGetErrorString(err));                                   \
      exit(EXIT_FAILURE);                                                 \
    }                                                                     \
  } while (0)

// Custom data types
struct MyInt4 { // 16 bytes
  int v0, v1, v2, v3;
};

// --- Kernels (Provided in the question) ---
template <typename T>
__global__ void pipeline_kernel_sync(T *global, uint64_t *clock, size_t copy_count) {
  extern __shared__ char s[];
  T *shared = reinterpret_cast<T *>(s);

  uint64_t clock_start = clock64();

  for (size_t i = 0; i < copy_count; ++i) {
    shared[blockDim.x * i + threadIdx.x] = global[blockDim.x * i + threadIdx.x];
  }

  uint64_t clock_end = clock64();

  // Only one thread needs to write this if we want total block time.
  // But the current atomicAdd sums up all thread's individual processing times.
  // To get an effective block time, we'll divide this sum by blockDim.x later.
  if (threadIdx.x == 0 && blockIdx.x == 0) { // Ensure only one block contributes if grid > 1
      atomicAdd(reinterpret_cast<unsigned long long *>(clock),
                clock_end - clock_start);
  }
}

template <typename T>
__global__ void pipeline_kernel_async(T *global, uint64_t *clock, size_t copy_count) {
  extern __shared__ char s[];
  T *shared = reinterpret_cast<T *>(s);
  uint64_t clock_start = clock64();
  for (size_t i = 0; i < copy_count; ++i) {
    __pipeline_memcpy_async(&shared[blockDim.x * i + threadIdx.x],
                            &global[blockDim.x * i + threadIdx.x], 
                            sizeof(T)); // Copy one element of type T per thread per iteration
  }
  __pipeline_commit();
  __pipeline_wait_prior(0); // Wait for all prior stages (0 means all previous stages)
  uint64_t clock_end = clock64();
  if (threadIdx.x == 0 && blockIdx.x == 0) { // Ensure only one block contributes if grid > 1
      atomicAdd(reinterpret_cast<unsigned long long *>(clock),
                clock_end - clock_start);
  }
}


// Helper function to run benchmark for a given type and configuration
template <typename T>
void run_test(const std::string& type_name, size_t element_size_bytes,
              size_t total_copy_bytes_per_block,
              int block_size, int repetitions, double gpu_clock_rate_ghz,
              std::vector<double>& sync_times_ms, std::vector<double>& async_times_ms,
              std::vector<double>& sync_bws, std::vector<double>& async_bws) {

    if (total_copy_bytes_per_block < block_size * element_size_bytes && total_copy_bytes_per_block % (block_size * element_size_bytes) != 0) {
        std::cout << std::setw(10) << type_name
                  << std::setw(12) << (total_copy_bytes_per_block / 1024.0)
                  << "  SKIPPED (total_copy_bytes < block_size * element_size or not multiple)" << std::endl;
        sync_times_ms.push_back(-1.0); async_times_ms.push_back(-1.0);
        sync_bws.push_back(-1.0); async_bws.push_back(-1.0);
        return;
    }
    
    size_t copy_count_per_thread = total_copy_bytes_per_block / (block_size * element_size_bytes);
    if (copy_count_per_thread == 0) {
         std::cout << std::setw(10) << type_name
                  << std::setw(12) << std::fixed << std::setprecision(2) << (total_copy_bytes_per_block / 1024.0)
                  << "  SKIPPED (copy_count_per_thread is 0)" << std::endl;
        sync_times_ms.push_back(-1.0); async_times_ms.push_back(-1.0);
        sync_bws.push_back(-1.0); async_bws.push_back(-1.0);
        return;
    }


    size_t num_elements_total = block_size * copy_count_per_thread; // Elements for one block
    size_t global_mem_size_bytes = num_elements_total * element_size_bytes;

    T* h_data = new T[num_elements_total];
    for(size_t i = 0; i < num_elements_total; ++i) {
        // Simple initialization
        char* p = reinterpret_cast<char*>(&h_data[i]);
        for(size_t b = 0; b < element_size_bytes; ++b) p[b] = (i+b) % 256;
    }

    T* d_global;
    uint64_t* d_clock;

    CUDA_CHECK(hipMalloc(&d_global, global_mem_size_bytes));
    CUDA_CHECK(hipMalloc(&d_clock, sizeof(uint64_t)));
    CUDA_CHECK(hipMemcpy(d_global, h_data, global_mem_size_bytes, hipMemcpyHostToDevice));

    dim3 grid_dim(1); // We are measuring per-block performance
    dim3 block_dim(block_size);
    size_t shared_mem_bytes = total_copy_bytes_per_block; // This is crucial

    // --- Sync Kernel ---
    double total_sync_cycles = 0;
    uint64_t h_clock_val;

    // Warm-up
    CUDA_CHECK(hipMemset(d_clock, 0, sizeof(uint64_t)));
    pipeline_kernel_sync<T><<<grid_dim, block_dim, shared_mem_bytes>>>(d_global, d_clock, copy_count_per_thread);
    CUDA_CHECK(hipDeviceSynchronize());

    for (int i = 0; i < repetitions; ++i) {
        CUDA_CHECK(hipMemset(d_clock, 0, sizeof(uint64_t)));
        pipeline_kernel_sync<T><<<grid_dim, block_dim, shared_mem_bytes>>>(d_global, d_clock, copy_count_per_thread);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(&h_clock_val, d_clock, sizeof(uint64_t), hipMemcpyDeviceToHost));
        total_sync_cycles += h_clock_val;
    }
    double avg_sync_cycles = static_cast<double>(total_sync_cycles) / repetitions;
    // The kernel's atomicAdd sums cycles from all threads. To get effective block time in cycles,
    // we should ideally take the max over threads.
    // Since only thread 0 writes, this is clock_end - clock_start for thread 0.
    // This is the time for thread 0 to complete its loops.
    double sync_time_ms = (avg_sync_cycles / (gpu_clock_rate_ghz * 1e9)) * 1000.0;
    double sync_bw_gbs = (total_copy_bytes_per_block / (sync_time_ms / 1000.0)) / (1024.0 * 1024.0 * 1024.0);
    
    sync_times_ms.push_back(sync_time_ms);
    sync_bws.push_back(sync_bw_gbs);

    // --- Async Kernel ---
    uint64_t total_async_cycles = 0;
    // Warm-up
    CUDA_CHECK(hipMemset(d_clock, 0, sizeof(uint64_t)));
    pipeline_kernel_async<T><<<grid_dim, block_dim, shared_mem_bytes>>>(d_global, d_clock, copy_count_per_thread);
    CUDA_CHECK(hipDeviceSynchronize());

    for (int i = 0; i < repetitions; ++i) {
        CUDA_CHECK(hipMemset(d_clock, 0, sizeof(uint64_t)));
        pipeline_kernel_async<T><<<grid_dim, block_dim, shared_mem_bytes>>>(d_global, d_clock, copy_count_per_thread);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(&h_clock_val, d_clock, sizeof(uint64_t), hipMemcpyDeviceToHost));
        // printf("cycle: %llf\n", h_clock_val);
        total_async_cycles += h_clock_val;
    }
    double avg_async_cycles = static_cast<double>(total_async_cycles) / repetitions;
    double async_time_ms = (avg_async_cycles / (gpu_clock_rate_ghz * 1e9)) * 1000.0;
    double async_bw_gbs = (total_copy_bytes_per_block / (async_time_ms / 1000.0)) / (1024.0 * 1024.0 * 1024.0);

    async_times_ms.push_back(async_time_ms);
    async_bws.push_back(async_bw_gbs);

    // Cleanup
    CUDA_CHECK(hipFree(d_global));
    CUDA_CHECK(hipFree(d_clock));
    delete[] h_data;

    // Print individual row
    std::cout << std::fixed << std::setprecision(2);
    std::cout << std::setw(10) << type_name
              << std::setw(12) << (total_copy_bytes_per_block / 1024.0) // KB
              << std::setw(15) << sync_time_ms
              << std::setw(15) << async_time_ms
              << std::setw(15) << sync_bw_gbs
              << std::setw(15) << async_bw_gbs
              << std::endl;
}


int main() {
    int device_id;
    CUDA_CHECK(hipGetDevice(&device_id));
    hipDeviceProp_t props;
    CUDA_CHECK(hipGetDeviceProperties(&props, device_id));
    double gpu_clock_rate_ghz = static_cast<double>(props.clockRate) / (1000.0 * 1000.0); // kHz to GHz

    std::cout << "GPU: " << props.name << std::endl;
    std::cout << "GPU Clock Rate: " << std::fixed << std::setprecision(2) << gpu_clock_rate_ghz << " GHz" << std::endl;
    std::cout << "Max Shared Memory Per Block: " << props.sharedMemPerBlock / 1024 << " KB" << std::endl;
    
    // Note: Max dynamic shared memory is props.sharedMemPerBlock - static shared memory used by kernel.
    // We are using dynamic shared memory.
    // On some architectures, you can opt-in for more shared memory per SM (e.g. 96KB vs 32KB cache on Ampere)
    // This might require hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, size);
    // For simplicity, we'll stick to standard limits.

    int block_size = 256; // A common block size
    int repetitions = /*for fast 100*/1; // Number of times to run each kernel for averaging
    std::cout << "Block Size: " << block_size << " threads" << std::endl;
    std::cout << "Repetitions per test: " << repetitions << std::endl << std::endl;

    std::cout << std::setw(10) << "Elem Type"
              << std::setw(12) << "Copy/Blk(KB)"
              << std::setw(15) << "Sync Time(ms)"
              << std::setw(15) << "Async Time(ms)"
              << std::setw(15) << "Sync BW(GB/s)"
              << std::setw(15) << "Async BW(GB/s)"
              << std::endl;
    std::cout << std::string(82, '-') << std::endl;

    std::vector<size_t> copy_sizes_bytes;
    // From 512 B up to 48 KB
    // std::vector<int> size = { 1, 2, 4, 8, 16, 32, 48};
    for (size_t kb_size : {/* for fast ncu1, 2, 4, 8, 16, 32,*/ 48}) { // KB values
         if (kb_size * 1024 <= props.sharedMemPerBlock) { // Ensure we don't exceed device limits
            copy_sizes_bytes.push_back(static_cast<size_t>(kb_size * 1024));
         } else if (!copy_sizes_bytes.empty() && copy_sizes_bytes.back() < props.sharedMemPerBlock && kb_size * 1024 > props.sharedMemPerBlock) {
            // Add the max possible if we jumped over it
            copy_sizes_bytes.push_back(props.sharedMemPerBlock);
         }
    }
    // Ensure the absolute max shared memory is tested if not already included and it's a power of 2 or common value.
    // For simplicity, we will use the list above. If props.sharedMemPerBlock is e.g. 49152 (48KB), it will be included.

    // Store all results for potential later processing if needed
    std::vector<double> all_sync_times, all_async_times, all_sync_bws, all_async_bws;

    // Test with int (4 Bytes)
    for (size_t cs : copy_sizes_bytes) {
        run_test<int>("int (4B)", sizeof(int), cs, block_size, repetitions, gpu_clock_rate_ghz,
                      all_sync_times, all_async_times, all_sync_bws, all_async_bws);
    }
    std::cout << std::string(82, '-') << std::endl;

    // Test with long long (8 Bytes)
    for (size_t cs : copy_sizes_bytes) {
        run_test<long long>("llong (8B)", sizeof(long long), cs, block_size, repetitions, gpu_clock_rate_ghz,
                            all_sync_times, all_async_times, all_sync_bws, all_async_bws);
    }
    std::cout << std::string(82, '-') << std::endl;

    // Test with MyInt4 (16 Bytes)
    for (size_t cs : copy_sizes_bytes) {
        run_test<MyInt4>("MyInt4(16B)", sizeof(MyInt4), cs, block_size, repetitions, gpu_clock_rate_ghz,
                         all_sync_times, all_async_times, all_sync_bws, all_async_bws);
    }
    std::cout << std::string(82, '-') << std::endl;

    return 0;
}
