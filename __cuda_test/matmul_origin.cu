#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib> // For rand() and srand()
#include <ctime>   // For time()
#include <cmath>   // For fabs()
#include <omp.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
// CUDA运行时API
#include <hip/hip_runtime.h>

// 用于检查CUDA API调用错误的宏
#define CUDA_CHECK(err)                                                               \
    do {                                                                              \
        hipError_t err_ = (err);                                                     \
        if (err_ != hipSuccess) {                                                    \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << ": "      \
                      << hipGetErrorString(err_) << std::endl;                       \
            exit(EXIT_FAILURE);                                                       \
        }                                                                             \
    } while (0)

static constexpr int  N = 64 * 50;
static constexpr int TILE_DIM = 32;
// CUDA Kernel
// 计算 C = A * B，其中 A, B, C 都是 N x N 的方阵
// 每个线程计算输出矩阵 C 的一个元素
// 
// 

// A[N, TILE_DIM] B[TILE_DIM, N]
// __global__ void sharedABMultiply(float *a, float* b, float *c)
// {
//     __shared__ float aTile[TILE_DIM][TILE_DIM],
//                      bTile[TILE_DIM][TILE_DIM];
//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     int col = blockIdx.x * blockDim.x + threadIdx.x;
//     float sum = 0.0f;
//     aTile[threadIdx.y][threadIdx.x] = a[row*TILE_DIM+threadIdx.x];
//     bTile[threadIdx.y][threadIdx.x] = b[threadIdx.y*N+col];
//     __syncthreads();
//     for (int i = 0; i < TILE_DIM; i++) {
//         sum += aTile[threadIdx.y][i]* bTile[i][threadIdx.x];
//         sum += aTile[threadIdx.y][i]* bTile[i][threadIdx.x];
//     }
//     c[row*N+col] = sum;
// }

__global__ void sharedABMultiply(float *a, float* b, float *c)
{
    __shared__ float aTile[TILE_DIM][TILE_DIM];
    __shared__ float bTile[TILE_DIM][TILE_DIM];
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float sum = 0.0f;
    aTile[threadIdx.y][threadIdx.x] = a[row * TILE_DIM + threadIdx.x];
    bTile[threadIdx.y][threadIdx.x] = b[threadIdx.y * N + col];
    __syncthreads();
    for(int i = 0; i < TILE_DIM; i++){
        sum += aTile[threadIdx.y][i] * bTile[i][threadIdx.x];
    }
    c[row * N + col] = sum;
}
__global__ void simpleMultiply(float *a, float* b, float *c)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;
    for (int i = 0; i < TILE_DIM; i++) {
        sum += a[row * TILE_DIM + i] * b[i * N + col];
    }
    c[row * N + col] = sum;
}


__global__ void coalescedMultiply(float *a, float* b, float *c)
{
    __shared__ float aTile[TILE_DIM * TILE_DIM];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    aTile[threadIdx.y * TILE_DIM + threadIdx.x] = a[row * TILE_DIM + threadIdx.x];
    // if(threadIdx.y == 0 && blockIdx.y == 0 && blockIdx. == 0) {
    //     printf("%d %d\n", threadIdx.x, threadIdx.y * TILE_DIM + threadIdx.x);
    // }
    __syncwarp();
    for (int i = 0; i < TILE_DIM; i++) {
        sum += aTile[threadIdx.y * TILE_DIM + i]* b[i*N+col];
    }
    c[row*N+col] = sum;
}


// 用于在CPU上执行矩阵乘法以验证结果的辅助函数
void matrixMultiplyCPU(const thrust::host_vector<float>& a, const thrust::host_vector<float>& b, thrust::host_vector<float>& c) {
    #pragma omp parallel for
    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col) {
            float sum = 0.0f;
            for (int i = 0; i < TILE_DIM; ++i) {
                sum += a[row * TILE_DIM + i] * b[i * N + col];
            }
            c[row * N + col] = sum;
        }
    }
}



void matrixTransMultiplyCPU(const thrust::host_vector<float>& a, thrust::host_vector<float>& c) {
    #pragma omp parallel for
    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col) {
            float sum = 0.0f;
            for (int i = 0; i < TILE_DIM; ++i) {
                sum += a[row * TILE_DIM + i] * a[col * TILE_DIM + i];
            }
            c[row * N + col] = sum;
        }
    }
}

__global__ void simpleTransMultiply(float *a, float *c, int M)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    for (int i = 0; i < TILE_DIM; i++) {
        sum += a[row*TILE_DIM+i] * a[col*TILE_DIM+i];
    }
    c[row*M+col] = sum;
}

__global__ void coalescedTransMultiply(float *a, float *c, int M)
{
    __shared__ float aTile[TILE_DIM][TILE_DIM],
                     transposedTile[TILE_DIM][TILE_DIM];
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    aTile[threadIdx.y][threadIdx.x] = a[row*TILE_DIM+threadIdx.x];
    transposedTile[threadIdx.x][threadIdx.y] =
        a[(blockIdx.x*blockDim.x + threadIdx.y)*TILE_DIM +
        threadIdx.x];
    __syncthreads();
    for (int i = 0; i < TILE_DIM; i++) {
        sum += aTile[threadIdx.y][i]* transposedTile[i][threadIdx.x];
    }
    c[row*M+col] = sum;
}

__global__ void coalescedPadTransMultiply(float *a, float *c, int M)
{
    __shared__ float aTile[TILE_DIM][TILE_DIM],
                     transposedTile[TILE_DIM][TILE_DIM + 1];
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    aTile[threadIdx.y][threadIdx.x] = a[row*TILE_DIM+threadIdx.x];
    transposedTile[threadIdx.x][threadIdx.y] =
        a[(blockIdx.x*blockDim.x + threadIdx.y)*TILE_DIM +
        threadIdx.x];
    __syncthreads();
    for (int i = 0; i < TILE_DIM; i++) {
        sum += aTile[threadIdx.y][i]* transposedTile[i][threadIdx.x];
    }
    c[row*M+col] = sum;
}
__global__ void coalescedPadSlowTransMultiply(float *a, float *c, int M)
{
    __shared__ float aTile[TILE_DIM][TILE_DIM + 1],
                     transposedTile[TILE_DIM][TILE_DIM + 1];
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    aTile[threadIdx.y][threadIdx.x] = a[row*TILE_DIM+threadIdx.x];
    transposedTile[threadIdx.x][threadIdx.y] =
        a[(blockIdx.x*blockDim.x + threadIdx.y)*TILE_DIM +
        threadIdx.x];
    __syncthreads();
    for (int i = 0; i < TILE_DIM; i++) {
        sum += aTile[threadIdx.y][i]* transposedTile[i][threadIdx.x];
    }
    c[row*M+col] = sum;
}

// __global__ void sharedTransMultiply(float *a, float *c, int M)
// {
//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     int col = blockIdx.x * blockDim.x + threadIdx.x;
//     __shared__ float aTile[TILE_DIM][TILE_DIM];
//     __shared__ float bTile[TILE_DIM]
    
//     float sum = 0.0f;
//     for (int i = 0; i < TILE_DIM; i++) {
//         sum += a[row*TILE_DIM+i] * a[col*TILE_DIM+i];
//     }
//     c[row*M+col] = sum;
// }


int main() {
    const int matrix_size_bytes = N * N * sizeof(float);

    thrust::host_vector<float> h_a(N * TILE_DIM);
    thrust::host_vector<float> h_b(TILE_DIM * N);
    thrust::host_vector<float> h_c_gpu(N * N); // 用于存储GPU计算结果
    thrust::host_vector<float> h_c_cpu(N * N); // 用于存储CPU计算结果 (验证用)

    // 用随机数初始化矩阵 A 和 B
    srand(static_cast<unsigned int>(time(0)));
    for (int i = 0; i < TILE_DIM * N; ++i) {
        // h_a[i] = 1;
        // h_b[i] = 1;
        h_a[i] = static_cast<float>(rand()) / RAND_MAX;
        h_b[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // --- 2. 将数据转移到设备 ---
    thrust::device_vector<float> dv_a = h_a;  // 自动拷贝到GPU
    thrust::device_vector<float> dv_b = h_b;
    thrust::device_vector<float> dv_c(N * N);

    // --- 3. 获取原始指针用于kernel调用 ---
    float* d_a = thrust::raw_pointer_cast(dv_a.data());
    float* d_b = thrust::raw_pointer_cast(dv_b.data());
    float* d_c = thrust::raw_pointer_cast(dv_c.data());
    // CUDA_CHECK(hipMalloc((void**)&d_a, ab_size_byts));
    // CUDA_CHECK(hipMalloc((void**)&d_b, ab_size_byts));
    // CUDA_CHECK(hipMalloc((void**)&d_c, matrix_size_bytes));

    // --- 3. 将数据从主机复制到设备 ---
    // CUDA_CHECK(hipMemcpy(d_a, h_a.data(), ab_size_byts, hipMemcpyHostToDevice));
    // CUDA_CHECK(hipMemcpy(d_b, h_b.data(), ab_size_byts, hipMemcpyHostToDevice));

    // --- 4. Kernel启动配置 ---
    // 定义每个block的线程数。通常选择16x16=256或32x32=1024个线程
    // 这里使用16x16，因为这是很多GPU架构上性能较好的选择
    dim3 blockSize(TILE_DIM, TILE_DIM);

    // 计算grid的维度，确保覆盖整个N x N矩阵
    // (N + D - 1) / D 是一种向上取整的常用方法
    dim3 gridSize(N /  TILE_DIM, N / TILE_DIM);

    std::cout << "Matrix dimensions: " << N << "x" << N << std::endl;
    std::cout << "Block dimensions: " << blockSize.x << "x" << blockSize.y << std::endl;
    std::cout << "Grid dimensions: " << gridSize.x << "x" << gridSize.y << std::endl;
    std::cout << "Total threads: " << gridSize.x * gridSize.y * blockSize.x * blockSize.y << std::endl;

    // --- 5. 执行Kernel ---
    std::cout << "Launching CUDA kernel..." << std::endl;
    simpleMultiply<<<gridSize, blockSize>>>(d_a, d_b, d_c);
    coalescedMultiply<<<gridSize, blockSize>>>(d_a, d_b, d_c);
    sharedABMultiply<<<gridSize, blockSize>>>(d_a, d_b, d_c);

    // 检查Kernel启动是否有错误
    CUDA_CHECK(hipGetLastError());
    // 同步设备，确保Kernel执行完毕
    CUDA_CHECK(hipDeviceSynchronize());
    std::cout << "CUDA kernel finished." << std::endl; 

    // return 0;
    // --- 6. 将结果从设备复制回主机 ---
    CUDA_CHECK(hipMemcpy(h_c_gpu.data(), d_c, matrix_size_bytes, hipMemcpyDeviceToHost));

    // --- 7. 验证结果 (可选但推荐) ---
    std::cout << "Verifying results on CPU..." << std::endl;
    matrixMultiplyCPU(h_a, h_b, h_c_cpu);

    bool match = true;
    float tolerance= 1e-4; // 浮点数比较容差
    int cnt = 0;
    std::cout << "compare here...";
    for (int i = 0; i < N * N; ++i) {
        if (std::fabs(h_c_gpu[i] - h_c_cpu[i]) > tolerance) {
            std::cerr << "Mismatch at index " << i / N << "," << i % N << ": GPU=" << h_c_gpu[i]
                      << ", CPU=" << h_c_cpu[i] << std::endl;
            match = false;
            cnt++;
            // break;
        }
        if(cnt > 10) break;
    }

    if (match) {
        std::cout << "Results match!" << std::endl;
    } else {
        std::cout << "Results DO NOT match!" << std::endl;
    }
    
    /**--------------------------------------------------------------trans */
    simpleTransMultiply<<<gridSize, blockSize>>>(d_a, d_c, N);
    coalescedTransMultiply<<<gridSize, blockSize>>>(d_a, d_c, N);
    coalescedPadTransMultiply<<<gridSize, blockSize>>>(d_a, d_c, N);
    coalescedPadSlowTransMultiply<<<gridSize, blockSize>>>(d_a, d_c, N);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

        std::cout << "CUDA kernel finished." << std::endl; 

    // return 0;
    // --- 6. 将结果从设备复制回主机 ---
    CUDA_CHECK(hipMemcpy(h_c_gpu.data(), d_c, matrix_size_bytes, hipMemcpyDeviceToHost));

    // --- 7. 验证结果 (可选但推荐) ---
    std::cout << "Verifying results on CPU..." << std::endl;
    matrixTransMultiplyCPU(h_a, h_c_cpu);

    match = true;
    tolerance= 1e-4; // 浮点数比较容差
    cnt = 0;
    std::cout << "compare here...";
    for (int i = 0; i < N * N; ++i) {
        if (std::fabs(h_c_gpu[i] - h_c_cpu[i]) > tolerance) {
            std::cerr << "Mismatch at index " << i / N << "," << i % N << ": GPU=" << h_c_gpu[i]
                      << ", CPU=" << h_c_cpu[i] << std::endl;
            match = false;
            cnt++;
            // break;
        }
        if(cnt > 10) break;
    }

    if (match) {
        std::cout << "Results match!" << std::endl;
    } else {
        std::cout << "Results DO NOT match!" << std::endl;
    }




    std::cout << "Program finished." << std::endl;
    return 0;
}
